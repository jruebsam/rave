#include "State.h"

State::State(){}

State::State(int nx_, int ny_)
:nx(nx_), ny(ny_)
{
    host.T = xt::zeros<float>({nx, ny});

    hipMalloc(&(current.T), nx*ny*sizeof(float)); 
    hipMalloc(&(target.T), nx*ny*sizeof(float)); 
}

void State::toDevice(){
    hipMemcpy(&(host.T), current.T, nx*ny*sizeof(float), hipMemcpyHostToDevice);
}

void State::toHost(){
    hipMemcpy(current.T, &(host.T), nx*ny*sizeof(float), hipMemcpyHostToDevice);
}

State::~State()
{
    hipFree(current.T);
    hipFree(target.T);
}