#include "hip/hip_runtime.h"
#include "Simulation.h"
#include <stdint.h>

__global__ void kernel(hipSurfaceObject_t surface, double time, double width, double height)
{

    const unsigned int IDx = blockIdx.x * blockDim.x +  threadIdx.x;
    const unsigned int IDy = blockIdx.y * blockDim.y + threadIdx.y;

    float x = IDx/width;
    float y = IDy/height;

    float v = cos(10*x)*sin(10*y)*cos(time)*0.5 + 0.5;
    uint8_t r, g, b; 

    float a=(1-v)/0.25;	
    int x0 = floor(a);
    int y0 = floor(255*(a - x0));

    switch(x0)
    {
        case 0: r=255;    g=y0;     b=0;   break;
        case 1: r=255-y0; g=255;    b=0;   break;
        case 2: r=0;      g=255;    b=y0;  break;
        case 3: r=0;      g=255-y0; b=255; break;
        case 4: r=0;      g=0;      b=255; break;
    }

    uchar4 data = make_uchar4(r, g, b, 0xff);
    surf2Dwrite(data, surface, IDx*sizeof(uchar4), IDy);
}

Simulation::Simulation(GLuint &texId_handle, int width_, int height_)
: width(width_), height(height_)
{
    hipGraphicsGLRegisterImage(&texRes, texId_handle, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
}


void Simulation::Step()
{
    hipGraphicsMapResources(1, &texRes);
    hipGraphicsSubResourceGetMappedArray(&data, texRes, 0, 0);

    hipResourceDesc resoureDescription;
    hipSurfaceObject_t surface = 0;

    memset(&resoureDescription, 0, sizeof(resoureDescription));
    resoureDescription.resType = hipResourceTypeArray;   
    resoureDescription.res.array.array = data; 

    hipCreateSurfaceObject(&surface, &resoureDescription);


    dim3 grids(30, 30);
    dim3 threads(16, 16);
    kernel<<<grids, threads>>>(surface, counter, width, height);

    hipGraphicsUnmapResources(1, &texRes);
    hipDestroySurfaceObject(surface);
    counter += 0.001;
}

Simulation::~Simulation()
{
    hipGraphicsUnregisterResource(texRes);
}
